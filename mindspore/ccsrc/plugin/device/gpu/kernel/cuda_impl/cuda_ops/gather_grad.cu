#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2024 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iostream>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/gather_grad.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/complex.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T>
using Complex = mindspore::utils::Complex<T>;

template <typename T, typename S>
__global__ void GatherGradKernel(const T *index, const S *grad, S *output, size_t dim_before_axis_index,
                                 size_t dim_at_axis_index, size_t dim_after_axis_index, size_t dim_at_axis_out,
                                 size_t dim_after_axis_out, size_t num) {
  for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < num; id += blockDim.x * gridDim.x) {
    T j = index[id];
    if (j < 0) {
      j += static_cast<T>(dim_at_axis_out);
    }
    CUDA_KERNEL_ASSERT(j >= 0);
    size_t j_read = static_cast<size_t>(j);
    CUDA_KERNEL_ASSERT(j_read < dim_at_axis_out);

    size_t offset = id % dim_after_axis_index + j_read * dim_after_axis_out +
                    ((id / (dim_after_axis_index * dim_at_axis_index)) % dim_before_axis_index) *
                      (dim_at_axis_out * dim_after_axis_out);
    MsAtomicAdd(output + offset, grad[id]);
  }
  return;
}

template <typename T, typename S>
hipError_t GatherGrad(const T *index, const S *grad, S *output, size_t dim_before_axis_index, size_t dim_at_axis_index,
                       size_t dim_after_axis_index, size_t dim_at_axis_out, size_t dim_after_axis_out, size_t num,
                       hipStream_t stream) {
  GatherGradKernel<<<GET_BLOCKS(num), GET_THREADS, 0, stream>>>(index, grad, output, dim_before_axis_index,
                                                                dim_at_axis_index, dim_after_axis_index,
                                                                dim_at_axis_out, dim_after_axis_out, num);
  return GetCudaStatus();
}

#define SPECIALIZE_KERNEL(T, S)                                                                       \
  template CUDA_LIB_EXPORT hipError_t GatherGrad<T, S>(                                              \
    const T *index, const S *grad, S *output, size_t dim_before_axis_index, size_t dim_at_axis_index, \
    size_t dim_after_axis_index, size_t dim_at_axis_out, size_t dim_after_axis_out, size_t num, hipStream_t stream);

SPECIALIZE_KERNEL(int, Complex<double>)
SPECIALIZE_KERNEL(int64_t, Complex<double>)
SPECIALIZE_KERNEL(int, Complex<float>)
SPECIALIZE_KERNEL(int64_t, Complex<float>)
SPECIALIZE_KERNEL(int, double)
SPECIALIZE_KERNEL(int64_t, double)
SPECIALIZE_KERNEL(int, float)
SPECIALIZE_KERNEL(int64_t, float)
SPECIALIZE_KERNEL(int, half)
SPECIALIZE_KERNEL(int64_t, half)
SPECIALIZE_KERNEL(int, int)
SPECIALIZE_KERNEL(int64_t, int)
SPECIALIZE_KERNEL(int, int8_t)
SPECIALIZE_KERNEL(int64_t, int8_t)
SPECIALIZE_KERNEL(int, int16_t)
SPECIALIZE_KERNEL(int64_t, int16_t)
SPECIALIZE_KERNEL(int, int64_t)
SPECIALIZE_KERNEL(int64_t, int64_t)
SPECIALIZE_KERNEL(int, unsigned char)
SPECIALIZE_KERNEL(int64_t, unsigned char)
SPECIALIZE_KERNEL(int, uint64_t)
SPECIALIZE_KERNEL(int64_t, uint64_t)
SPECIALIZE_KERNEL(int, uint32_t)
SPECIALIZE_KERNEL(int64_t, uint32_t)
SPECIALIZE_KERNEL(int, uint16_t)
SPECIALIZE_KERNEL(int64_t, uint16_t)
SPECIALIZE_KERNEL(int, bool)
SPECIALIZE_KERNEL(int64_t, bool)

#undef SPECIALIZE_KERNEL
